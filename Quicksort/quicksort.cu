#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32
#define N 8

void gpu_qsort(int *data, int nitems);

__global__ void gpu_qsort_Kernel(int *arr, int left, int right, int depth){

    hipStream_t sl,sr;

    int *lptr = arr + left;
    int *rptr = arr + right;
    int pivot = arr[(left+right)/2];

    int lval, rval;
    int nright, nleft;

    while (lptr <= rptr){
        lval = *lptr;
        rval = *rptr;

        // move elements smaller than the pivot value to left subarray
        while (lval < pivot && lptr < arr+right){
            lptr++;
            lval = *lptr;
        }

	// move elements larger than the pivot value to right subarray
        while (rval > pivot && rptr > arr+left){
            rptr--;
            rval = *rptr;
        }

        // swap
        if (lptr <= rptr){
            *lptr = rval;
            *rptr = lval;
            lptr++;
            rptr--;
        }
    }

    nright = rptr - arr;
    nleft  = lptr - arr;

    // Launch a new block to sort the left part.
    if (left < nright){
        hipStreamCreateWithFlags(&sl, hipStreamNonBlocking);
        gpu_qsort_Kernel<<< 1, 1, 0, sl >>>(arr, left, nright, depth+1);
        hipStreamDestroy(sl);
    }

    // Launch a new block to sort the right part.
    if (nleft < right){
        hipStreamCreateWithFlags(&sr, hipStreamNonBlocking);
        gpu_qsort_Kernel<<< 1, 1, 0, sr >>>(arr, nleft, right, depth+1);
        hipStreamDestroy(sr);
    }
}


void gpu_qsort(int *data, int n){
    int* temp;
    int left = 0;
    int right = n-1;
    int block_size = 1;
    int threads_per_block = 10;

    // set the max depth
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    hipMalloc((void**)&temp, n*sizeof(int));

    hipMemcpy(temp, data, n*sizeof(int), hipMemcpyHostToDevice);

    gpu_qsort_Kernel <<< block_size, threads_per_block >>> (temp, left, right, 0);

    hipDeviceSynchronize();

    hipMemcpy(data, temp, n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(temp);
    // clean up the driver 
    hipDeviceReset();
}


int main(int argc, char* argv[]){
	int n; //problem size
	FILE* fp;
	FILE* fp_out;
	char outname[100];

	fp = fopen(argv[1], "r");

        fscanf(fp, "%d", &n);

        int* inputarr = (int*)malloc(n*sizeof(int));

        for(int i = 0; i < n; i++){
        	fscanf(fp, "%d", &inputarr[i]);
        }

	gpu_qsort(inputarr, n);

	sprintf(outname, "%d", n);
        strcat(outname, ".txt");
	fp_out = fopen(outname, "w");

	for(int i = 0; i < n; i++) fprintf(fp_out, "%d ", inputarr[i]);

}
