#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void matrixMultiplication(float *A, float *B, float *Result, int Width);
__global__ void matrixMulKernel(float *dA, float *dB, float *dResult, int Width);

int main(int argc, char *argv[])
{
    int N = atoi(argv[1]);
 
    float *A, *B, *Result;

    A = (float *)calloc(N * N, sizeof(float));
    B = (float *)calloc(N * N, sizeof(float));
    Result = (float *)calloc(N * N, sizeof(float));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = 1;
            B[i * N + j] = 2;
        }
    }

    matrixMultiplication(A, B, Result, N);

    free(A);
    free(B);
    free(Result);
}

void matrixMultiplication(float *A, float *B, float *Result, int Width)
{
    int size = sizeof(float) * Width * Width;
    float *dA, *dB, *dResult;

    hipMalloc(&dA, size);
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);

    hipMalloc(&dB, size);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    hipMalloc(&dResult, size);

    dim3 dimBlock(Width, Width);
    dim3 dimGrid(Width/dimBlock.x, Width/dimBlock.y);
    //dim3 dimGrid(1, 1);

    matrixMulKernel<<<dimGrid, dimBlock>>>(dA, dB, dResult, Width);

    hipMemcpy(Result, dResult, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < Width; i++)
    {
        for (int j = 0; j < Width; j++)
        {
            printf("%f   ", Result[i * Width + j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dResult);
}

__global__ void matrixMulKernel(float *dA, float *dB, float *dResult, int width)
{

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;

    for (int i = 0; i < width; i++)
    {
        sum += dA[row * width + i] * dB[i * width + col];
    }
    dResult[row * width + col] = sum;
  
}