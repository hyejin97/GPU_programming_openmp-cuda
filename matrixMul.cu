#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void matrixMultiplication(float *A, float *B, float *Result, int Width);
__global__ void matrixMulKernel(float *dA, float *dB, float *dResult, int Width);
int minimum(int, int);

int main(int argc, char *argv[])
{
    int N = atoi(argv[1]);

    float *A, *B, *Result;

    A = (float *)calloc(N * N, sizeof(float));
    B = (float *)calloc(N * N, sizeof(float));
    Result = (float *)calloc(N * N, sizeof(float));

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = 1;
            B[i * N + j] = 2;
        }
    }

    matrixMultiplication(A, B, Result, N);

    free(A);
    free(B);
    free(Result);
}

void matrixMultiplication(float *A, float *B, float *Result, int Width)
{
    int size = sizeof(float) * Width * Width;
    float *dA, *dB, *dResult;

    hipMalloc(&dA, size);
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);

    hipMalloc(&dB, size);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    hipMalloc(&dResult, size);

    dim3 dimBlock(32, 32);
    dim3 dimGrid(ceil(double(Width + 31) / 32), ceil(double(Width + 31) / 32));
    if (Width > 10000)
    {
        dimGrid.x = ceil(double(Width + 31) / minimum(Width, 1000));
        dimGrid.y = ceil(double(Width + 31) / minimum(Width, 1000));
    }

    matrixMulKernel<<<dimGrid, dimBlock>>>(dA, dB, dResult, Width);

    hipMemcpy(Result, dResult, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < Width; i++)
    {
        for (int j = 0; j < Width; j++)
        {
            printf("%f   ", Result[i * Width + j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dResult);
}
int minimum(int n, int x)
{
    if (n < x)
    {
        return n;
    }
    else
    {
        return x;
    }
}
__global__ void matrixMulKernel(float *dA, float *dB, float *dResult, int width)
{

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int sum = 0;
    if (col < width && row < width)
    {
        for (int i = 0; i < width; i++)
        {
            sum += dA[row * width + i] * dB[i * width + col];
        }
        dResult[row * width + col] = sum;
    }
}
